// Perform FFTs using cuFFT. [docs](https://docs.nvidia.com/cuda/cufft/)

// We use the scatter functionality here for both cuFFT and
// VkFFT. The rest of this is for cuFFT only.

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <cstdio>
#include <cstdint>


// A minimal CUFFT error checker.
#ifndef CUFFT_CHECK
#define CUFFT_CHECK(call)                                                   \
  do {                                                                      \
    hipfftResult _e = (call);                                                \
    if (_e != HIPFFT_SUCCESS) {                                              \
      printf("CUFFT error %d at %s:%d\n", (int)_e, __FILE__, __LINE__);     \
    }                                                                       \
  } while (0)
#endif

struct PlanWrap {
    hipfftHandle plan_r2c;
    hipfftHandle plan_c2r;
    hipStream_t stream;
};

// https://docs.nvidia.com/cuda/cufft/#cufftplan3d
extern "C"
void* make_plan(int nx, int ny, int nz, void* cu_stream) {
    auto* w = new PlanWrap();

    w->stream = reinterpret_cast<hipStream_t>(cu_stream);

    // With Plan3D, Z is the fastest-changing dimension (contiguous); x is the slowest.
    CUFFT_CHECK(hipfftPlan3d(&w->plan_r2c, nx, ny, nz, HIPFFT_R2C));
    CUFFT_CHECK(hipfftPlan3d(&w->plan_c2r, nx, ny, nz, HIPFFT_C2R));

    CUFFT_CHECK(hipfftSetStream(w->plan_r2c, w->stream));
    CUFFT_CHECK(hipfftSetStream(w->plan_c2r, w->stream));

    return w;
}


extern "C"
void destroy_plan(void* plan) {
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;

    hipfftDestroy(w->plan_r2c);
    hipfftDestroy(w->plan_c2r);

    delete w;
}

// https://docs.nvidia.com/cuda/cufft/#cufftexecr2c-and-cufftexecd2z
// Performs a forward real-to-copmlex FFT of rho. Note: This is more efficient
// than complex-to-complex.
extern "C"
void exec_forward(void* plan, float* rho_real, hipfftComplex* rho) {
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;

    CUFFT_CHECK(hipfftExecR2C(w->plan_r2c, rho_real, rho));
}

extern "C"
void exec_inverse(
    void* plan,
    hipfftComplex* exk,
    hipfftComplex* eyk,
    hipfftComplex* ezk,
    float* ex,
    float* ey,
    float* ez
){
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;

    CUFFT_CHECK(hipfftExecC2R(w->plan_c2r, exk, ex));
    CUFFT_CHECK(hipfftExecC2R(w->plan_c2r, eyk, ey));
    CUFFT_CHECK(hipfftExecC2R(w->plan_c2r, ezk, ez));
}