// vk_fft.c
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>      // Driver API

#define VKFFT_BACKEND 1  // CUDA // todo: Probably not required, as set in build system.
#include "vkFFT.h"     // third-party library header (VkFFTApplication, etc.)
#include "vk_fft.h"    // your FFI header (prototypes above)


// typedef struct {
//     hipDevice_t  dev;
//     hipCtx_t ctx;
//     hipStream_t  stream;
//     int owns_stream; // 0 = adopted, 1 = created
// } VkContext;
//
// typedef struct {
//     VkFFTApplication app_r2c;
//     VkFFTApplication app_c2r;
//     VkFFTConfiguration cfg_r2c;
//     VkFFTConfiguration cfg_c2r;
//     uint64_t Nx, Ny, Nz;
// } VkFftPlan;

typedef struct VkContext {
    hipDevice_t  dev;
    hipCtx_t ctx;
    hipStream_t  stream;
    int       owns_stream;
} VkContext;

typedef struct VkFftPlan {
    VkFFTApplication   app;
    VkFFTConfiguration cfg;
    hipDevice_t           cu_dev;
    hipCtx_t          cu_ctx;
    hipStream_t       stream;
    uint64_t           Nx, Ny, Nz;   // store dims for caller
} VkFftPlan;

void* vk_make_context_from_stream(void* cu_stream_void) {
    VkContext* c = (VkContext*)calloc(1, sizeof(VkContext));
    if (!c) return NULL;

    c->stream = (hipStream_t)cu_stream_void;
    c->owns_stream = 0;

    hipInit(0);

    hipCtx_t cur = NULL;
    hipCtxGetCurrent(&cur);
    if (cur == NULL) {
        hipDevice_t dev0;
        hipDeviceGet(&dev0, 0);
        hipDevicePrimaryCtxRetain(&cur, dev0);
        hipCtxSetCurrent(cur);
    }

    c->ctx = cur;
    hipCtxGetDevice(&c->dev);
    return c;
}

void* vk_make_context_default(void) {
    VkContext* c = (VkContext*)calloc(1, sizeof(VkContext));
    if (!c) return NULL;

    hipInit(0);
    hipDeviceGet(&c->dev, 0);

    hipCtx_t primary = NULL;
    hipDevicePrimaryCtxRetain(&primary, c->dev);
    hipCtxSetCurrent(primary);
    c->ctx = primary;

    hipStreamCreateWithFlags(&c->stream, hipStreamDefault);
    c->owns_stream = 1;
    return c;
}

void vk_destroy_context(void* ctx_) {
    VkContext* c = (VkContext*)ctx_;
    if (!c) return;
    if (c->owns_stream) hipStreamDestroy(c->stream);
    if (c->owns_stream) hipDevicePrimaryCtxRelease(c->dev);
    free(c);
}

// 3D R2C/C2R
void* make_plan(void* ctx_, int32_t nx, int32_t ny, int32_t nz, void* cu_stream)
{
    VkContext* g = (VkContext*)ctx_;

    VkFftPlan* p = (VkFftPlan*)calloc(1, sizeof(VkFftPlan));
    if (!p) return NULL;

    p->cu_dev  = g->dev;
    p->cu_ctx  = g->ctx;
    p->stream  = cu_stream ? (hipStream_t)cu_stream : (hipStream_t)g->stream;

    p->Nx = (uint64_t)nx;
    p->Ny = (uint64_t)ny;
    p->Nz = (uint64_t)nz;

    VkFFTConfiguration* cfg = &p->cfg;
    memset(cfg, 0, sizeof(*cfg));

    cfg->device      = &p->cu_dev;
    cfg->context     = &p->cu_ctx;   // <-- important for CUDA backend
    cfg->stream      = &p->stream;
    cfg->num_streams = 1;

    cfg->isInputFormatted  = 1;
    cfg->isOutputFormatted = 1;

    cfg->FFTdim  = 3;
    cfg->size[0] = (uint64_t)nz;
    cfg->size[1] = (uint64_t)ny;
    cfg->size[2] = (uint64_t)nx;

    cfg->performR2C    = 1;
    cfg->normalize     = 0;
    cfg->numberBatches = 1;

    VkFFTResult res = initializeVkFFT(&p->app, *cfg);
    if (res != VKFFT_SUCCESS) {
        free(p);
        return NULL;
    }

    return p;
}

void destroy_plan(void* plan_) {
    VkFftPlan* p = (VkFftPlan*)plan_;
    if (!p) return;
    deleteVkFFT(&p->app);
    free(p);
}

void exec_forward(void* plan_, void* real_in, void* complex_out) {
    VkFftPlan* p = (VkFftPlan*)plan_;

    hipCtxSetCurrent(p->cu_ctx);  // <-- make sure we’re on the right ctx

    hipDeviceptr_t in  = (hipDeviceptr_t)real_in;
    hipDeviceptr_t out = (hipDeviceptr_t)complex_out;

    VkFFTLaunchParams lp;
    memset(&lp, 0, sizeof(lp));

    lp.buffer       = (void**)&in;
    lp.outputBuffer = (void**)&out;

    VkFFTResult res = VkFFTAppend(&p->app, -1, &lp);
    if (res != VKFFT_SUCCESS) {
        // printf("VkFFT forward failed: %d\n", res);
    }
}

void exec_inverse(void* plan_, void* complex_in, void* real_out) {
    VkFftPlan* p = (VkFftPlan*)plan_;

    hipCtxSetCurrent(p->cu_ctx);  // <-- same here

    hipDeviceptr_t in  = (hipDeviceptr_t)complex_in;
    hipDeviceptr_t out = (hipDeviceptr_t)real_out;

    VkFFTLaunchParams lp;
    memset(&lp, 0, sizeof(lp));

    lp.buffer       = (void**)&in;
    lp.outputBuffer = (void**)&out;

    VkFFTResult res = VkFFTAppend(&p->app, 1, &lp);
    if (res != VKFFT_SUCCESS) {
        // printf("VkFFT inverse failed: %d\n", res);
    }
}