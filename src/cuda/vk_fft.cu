// vk_fft.c
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>      // Driver API

#define VKFFT_BACKEND 1  // CUDA // todo: Probably not required, as set in build system.
#include "vkFFT.h"     // third-party library header (VkFFTApplication, etc.)
#include "vk_fft.h"    // your FFI header (prototypes above)


// typedef struct {
//     hipDevice_t  dev;
//     hipCtx_t ctx;
//     hipStream_t  stream;
//     int owns_stream; // 0 = adopted, 1 = created
// } VkContext;
//
// typedef struct {
//     VkFFTApplication app_r2c;
//     VkFFTApplication app_c2r;
//     VkFFTConfiguration cfg_r2c;
//     VkFFTConfiguration cfg_c2r;
//     uint64_t Nx, Ny, Nz;
// } VkFftPlan;


typedef struct VkContext {
    hipDevice_t  dev;
    hipCtx_t ctx;
    hipStream_t  stream;
    int       owns_stream; // 0 = adopted, 1 = we created it
} VkContext;

// Our plan: single app, single cfg, plus cached CUDA handles.
typedef struct VkFftPlan {
    VkFFTApplication  app;
    VkFFTConfiguration cfg;
    hipDevice_t           cu_dev;
    hipCtx_t          cu_ctx;
    hipStream_t       stream;
} VkFftPlan;



void* vk_make_context_from_stream(void* cu_stream_void) {
    VkContext* c = (VkContext*)calloc(1, sizeof(VkContext));
    if (!c) return NULL;

    c->stream = (hipStream_t)cu_stream_void;
    c->owns_stream = 0;

    hipInit(0);

    hipCtx_t cur = NULL;
    hipCtxGetCurrent(&cur);
    if (cur == NULL) {
        hipDevice_t dev0;
        hipDeviceGet(&dev0, 0);
        hipDevicePrimaryCtxRetain(&cur, dev0);
        hipCtxSetCurrent(cur);
    }

    c->ctx = cur;
    hipCtxGetDevice(&c->dev);
    return c;
}

void* vk_make_context_default(void) {
    VkContext* c = (VkContext*)calloc(1, sizeof(VkContext));
    if (!c) return NULL;

    hipInit(0);
    hipDeviceGet(&c->dev, 0);

    hipCtx_t primary = NULL;
    hipDevicePrimaryCtxRetain(&primary, c->dev);
    hipCtxSetCurrent(primary);
    c->ctx = primary;

    hipStreamCreateWithFlags(&c->stream, hipStreamDefault);
    c->owns_stream = 1;
    return c;
}

void vk_destroy_context(void* ctx_) {
    VkContext* c = (VkContext*)ctx_;
    if (!c) return;
    if (c->owns_stream) hipStreamDestroy(c->stream);
    if (c->owns_stream) hipDevicePrimaryCtxRelease(c->dev);
    free(c);
}

void* make_plan(void* ctx_, int32_t nx, int32_t ny, int32_t nz, void* cu_stream)
{
    VkContext* g = (VkContext*)ctx_;

    VkFftPlan* p = (VkFftPlan*)calloc(1, sizeof(VkFftPlan));
    if (!p) return NULL;

    p->cu_dev  = g->dev;
    p->cu_ctx  = g->ctx;
    p->stream  = cu_stream ? (hipStream_t)cu_stream
                           : (hipStream_t)g->stream;

    VkFFTConfiguration* cfg = &p->cfg;
    memset(cfg, 0, sizeof(*cfg));

    cfg->device      = &p->cu_dev;
    cfg->stream      = &p->stream;
    cfg->num_streams = 1;

    cfg->FFTdim  = 3;
    cfg->size[0] = (uint64_t)nz;
    cfg->size[1] = (uint64_t)ny;
    cfg->size[2] = (uint64_t)nx;

    cfg->performR2C    = 1;
    cfg->normalize     = 0;
    cfg->numberBatches = 1;

    VkFFTResult res = initializeVkFFT(&p->app, *cfg);
    if (res != VKFFT_SUCCESS) {
        free(p);
        return NULL;
    }

    return p;
}

void destroy_plan(void* plan_) {
    VkFftPlan* p = (VkFftPlan*)plan_;
    if (!p) return;
    deleteVkFFT(&p->app);
    free(p);
}

void exec_forward(void* plan_, void* real_in, void* complex_out) {
    VkFftPlan* p = (VkFftPlan*)plan_;

    hipDeviceptr_t in  = (hipDeviceptr_t)real_in;
    hipDeviceptr_t out = (hipDeviceptr_t)complex_out;

    VkFFTLaunchParams lp;
    memset(&lp, 0, sizeof(lp));

    lp.buffer       = (void**)&in;
    lp.outputBuffer = (void**)&out;
    // no lp.stream: VkFFTLaunchParams in your headers doesn’t have it; stream is in cfg.

    VkFFTAppend(&p->app, -1, &lp); // forward
}

void exec_inverse(void* plan_, void* complex_in, void* real_out) {
    VkFftPlan* p = (VkFftPlan*)plan_;

    hipDeviceptr_t in  = (hipDeviceptr_t)complex_in;
    hipDeviceptr_t out = (hipDeviceptr_t)real_out;

    VkFFTLaunchParams lp;
    memset(&lp, 0, sizeof(lp));

    lp.buffer       = (void**)&in;
    lp.outputBuffer = (void**)&out;

    VkFFTAppend(&p->app, 1, &lp); // inverse
}