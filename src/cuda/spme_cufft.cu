#include "hip/hip_runtime.h"
// Code here is only used by the cuFFT pipeline.

// We use the scatter functionality here for both cuFFT and
// VkFFT. The rest of this is for cuFFT only.

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <cstdio>
#include <cstdint>

#include "spme_shared.cu"


// A minimal CUFFT error checker.
#ifndef CUFFT_CHECK
#define CUFFT_CHECK(call)                                                   \
  do {                                                                      \
    hipfftResult _e = (call);                                                \
    if (_e != HIPFFT_SUCCESS) {                                              \
      printf("CUFFT error %d at %s:%d\n", (int)_e, __FILE__, __LINE__);     \
    }                                                                       \
  } while (0)
#endif

struct PlanWrap {
    hipfftHandle plan_r2c;
    hipfftHandle plan_c2r_many; // batch=3 for exk, eyk, ezk
    size_t n_real;  // nx*ny*nz
    size_t n_cmplx; // nx*ny*(nz/2+1) if using z as transform axis
    int nx, ny, nz;
    hipStream_t stream;
};

extern "C"
void* make_plan_r2c_c2r_many(int nx, int ny, int nz, void* cu_stream) {
    auto* w = new PlanWrap();
    w->nx = nx; w->ny = ny; w->nz = nz;
    w->n_real  = size_t(nx)*ny*nz;
    w->n_cmplx = size_t(nx)*ny*(nz/2 + 1);
    w->stream = reinterpret_cast<hipStream_t>(cu_stream);

    CUFFT_CHECK(hipfftPlan3d(&w->plan_r2c, nx, ny, nz, HIPFFT_R2C));
    CUFFT_CHECK(hipfftSetStream(w->plan_r2c, w->stream));

    // PlanMany for 3 fields back to real grids
    int n[3] = {nx, ny, nz};
    int inembed[3]  = {nx, ny, nz/2 + 1};
    int onembed[3]  = {nx, ny, nz};
    int istride = 1, ostride = 1;
    int idist = nx*ny*(nz/2 + 1);
    int odist = nx*ny*nz;
    int batch = 3;

    CUFFT_CHECK(hipfftPlanMany(&w->plan_c2r_many, 3, n,
                              inembed, istride, idist,
                              onembed, ostride, odist,
                              HIPFFT_C2R, batch));
    CUFFT_CHECK(hipfftSetStream(w->plan_c2r_many, w->stream));
    return w;
}

extern "C" __global__
void apply_ghat_and_grad(
    const hipfftComplex* __restrict__ rho,
    hipfftComplex* __restrict__ exk,
    hipfftComplex* __restrict__ eyk,
    hipfftComplex* __restrict__ ezk,
    const float* __restrict__ kx,
    const float* __restrict__ ky,
    const float* __restrict__ kz,
    const float* __restrict__ bx,
    const float* __restrict__ by,
    const float* __restrict__ bz,
    int nx, int ny, int nz, float vol, float alpha)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_cmplx = nx*ny*(nz/2 + 1);
    if (idx >= n_cmplx) return;

    int nxny = nx*ny;
    int iz = idx / nxny;          // 0 .. nz/2
    int rem = idx - iz*nxny;
    int iy = rem / nx;            // 0 .. ny-1
    int ix = rem - iy*nx;         // 0 .. nx-1

    float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];
    float k2  = fmaf(kxv, kxv, fmaf(kyv, kyv, kzv*kzv));
    if (k2 == 0.f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    float bmod2 = bx[ix] * by[iy] * bz[iz];
    if (bmod2 <= 1e-10f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    float ghat = (2.0f*3.14159265358979323846f*2.0f / vol) * __expf(-k2/(4.0f*alpha*alpha)) / (k2*bmod2);

    float a = rho[idx].x * ghat;
    float b = rho[idx].y * ghat;

    exk[idx].x =  kxv * b; exk[idx].y = -kxv * a;
    eyk[idx].x =  kyv * b; eyk[idx].y = -kyv * a;
    ezk[idx].x =  kzv * b; ezk[idx].y = -kzv * a;

    // after computing a,b and setting exk/eyk/ezk
    const bool rim_x = (ix==0) || ((nx%2)==0 && ix==(nx/2));
    const bool rim_y = (iy==0) || ((ny%2)==0 && iy==(ny/2));
    const bool rim_z = (iz==0) || ((nz%2)==0 && iz==(nz/2));
    // Imag parts must be zero on self-conjugate rims
    if (rim_x) exk[idx].y = 0.0f;
    if (rim_y) eyk[idx].y = 0.0f;
    if (rim_z) ezk[idx].y = 0.0f;
    // Optional (more conservative): zero the entire component on its Nyquist plane
    // if ((nx%2)==0 && ix==(nx/2)) exk[idx].x = exk[idx].y = 0.0f;
    // if ((ny%2)==0 && iy==(ny/2)) eyk[idx].x = eyk[idx].y = 0.0f;
    // if ((nz%2)==0 && iz==(nz/2)) ezk[idx].x = ezk[idx].y = 0.0f;
}

extern "C"
void destroy_plan_r2c_c2r_many(void* plan) {
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;
    hipfftDestroy(w->plan_r2c);
    hipfftDestroy(w->plan_c2r_many);
    delete w;
}

extern "C"
void apply_ghat_and_grad_launch(
    const void* rho,
    void* exk, void* eyk, void* ezk,
    const void* kx, const void* ky, const void* kz,
    const void* bx, const void* by, const void* bz,
    int nx, int ny, int nz, float vol, float alpha,
    void* cu_stream)
{
    auto s = reinterpret_cast<hipStream_t>(cu_stream);
    int n = nx*ny*(nz/2 + 1);          // <-- define n here

    int threads = 256;
    int blocks  = (n + threads - 1) / threads;

    apply_ghat_and_grad<<<blocks, threads, 0, s>>>(
        static_cast<const hipfftComplex*>(rho),
        static_cast<hipfftComplex*>(exk),
        static_cast<hipfftComplex*>(eyk),
        static_cast<hipfftComplex*>(ezk),
        static_cast<const float*>(kx),
        static_cast<const float*>(ky),
        static_cast<const float*>(kz),
        static_cast<const float*>(bx),
        static_cast<const float*>(by),
        static_cast<const float*>(bz),
        nx, ny, nz, vol, alpha
    );
}

extern "C"
void exec_inverse_ExEyEz_c2r(void* plan,
                                  hipfftComplex* exk, /* base of [exk|eyk|ezk] */
                                  hipfftComplex* /*eyk*/,
                                  hipfftComplex* /*ezk*/,
                                  float* ex /* base of [ex|ey|ez] */,
                                  float* /*ey*/,
                                  float* /*ez*/)
{
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;

    // exk must point to 3*n_cmplx contiguous hipfftComplex
    // ex  must point to 3*n_real   contiguous float
    CUFFT_CHECK(hipfftExecC2R(w->plan_c2r_many, exk, ex));
}

extern "C"
void spme_exec_forward_r2c(void* plan, float* rho_real, hipfftComplex* rho_k) {
    auto* w = reinterpret_cast<PlanWrap*>(plan);
    if (!w) return;
    CUFFT_CHECK(hipfftExecR2C(w->plan_r2c, rho_real, rho_k));
}

__global__ void energy_half_spectrum(
    const hipfftComplex* __restrict__ rho_k,
    const float* __restrict__ kx, const float* __restrict__ ky, const float* __restrict__ kz,
    const float* __restrict__ bx, const float* __restrict__ by, const float* __restrict__ bz,
    int nx, int ny, int nz, float vol, float alpha,
    double* __restrict__ out_partial)
{
    extern __shared__ double ssum[];
    int tid = threadIdx.x;
    double acc = 0.0;

    int nxy = nx*ny;
    int n_cmplx = nxy*(nz/2 + 1);

    // CUFFT normalization for |rho_k|^2
    int N = nx*ny*nz;
    double invN2 = 1.0 / (double(N) * double(N));

    for (int idx = blockIdx.x*blockDim.x + tid; idx < n_cmplx; idx += gridDim.x*blockDim.x) {
        int iz = idx / nxy;
        int rem = idx - iz*nxy;
        int iy = rem / nx;
        int ix = rem - iy*nx;

        float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];
        float k2  = fmaf(kxv,kxv, fmaf(kyv,kyv, kzv*kzv));
        if (k2 == 0.f) continue;

        float bmod2 = bx[ix]*by[iy]*bz[iz];
        if (bmod2 <= 1e-10f) continue;

        float ghat = (2.0f*3.14159265358979323846f*2.0f / vol) * __expf(-k2/(4.0f*alpha*alpha)) / (k2*bmod2);

        float a = rho_k[idx].x, b = rho_k[idx].y;
        double mag2 = double(a)*a + double(b)*b;

        int twice = (iz==0 || ((nz%2)==0 && iz==(nz/2))) ? 1 : 2;
        acc += 0.5 * double(twice) * double(ghat) * (mag2 * invN2);
    }
    ssum[tid] = acc;
    __syncthreads();
    for (int s = blockDim.x/2; s>0; s>>=1) {
        if (tid < s) ssum[tid] += ssum[tid+s];
        __syncthreads();
    }
    if (tid==0) out_partial[blockIdx.x] = ssum[0];
}

extern "C"
void energy_half_spectrum_launch(
    const void* rho_k,
    const void* kx, const void* ky, const void* kz,
    const void* bx, const void* by, const void* bz,
    int nx, int ny, int nz, float vol, float alpha,
    void* partial_sums,   // device ptr to double[blocks]
    int blocks, int threads, void* cu_stream)
{
    auto s = reinterpret_cast<hipStream_t>(cu_stream);
    size_t shmem = size_t(threads) * sizeof(double);

    energy_half_spectrum<<<blocks, threads, shmem, s>>>(
        static_cast<const hipfftComplex*>(rho_k),
        static_cast<const float*>(kx), static_cast<const float*>(ky), static_cast<const float*>(kz),
        static_cast<const float*>(bx), static_cast<const float*>(by), static_cast<const float*>(bz),
        nx, ny, nz, vol, alpha,
        static_cast<double*>(partial_sums));
}