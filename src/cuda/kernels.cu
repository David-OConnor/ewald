// Contains device code; GPU kernels. It does not perform FFTs. This does not include a cuFFT, nor vkFFT dependency.

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

// Note: We use float2 instead of cufftComplex, as it doesn't rely on cuFFT.

__device__ __forceinline__
int wrap(int a, int n) {
    a %= n; return (a < 0) ? a + n : a;
}


//  Corresponds directly to a host function.
__device__
void bspline4_weights(float s, int* i0, float w[4]) {
    float sfloor = floorf(s);
    float u = s - sfloor;
    *i0 = (int)sfloor - 1;

    float u2 = u * u;
    float u3 = fmaf(u2, u, 0.0f);

    float w0 = (1.0f - u);
    w0 = (w0 * w0 * w0) * (1.0f / 6.0f);

    float w1 = (3.0f * u3 - 6.0f * u2 + 4.0f) * (1.0f / 6.0f);
    float w2 = (-3.0f * u3 + 3.0f * u2 + 3.0f * u + 1.0f) * (1.0f / 6.0f);
    float w3 = u3 * (1.0f / 6.0f);

    w[0] = w0;
    w[1] = w1;
    w[2] = w2;
    w[3] = w3;
}

// Kernel for charge spreading. Corresponds directly to a host function.
extern "C" __global__
void spread_charges(
    const float3* pos,
    const float*  q,
    float* rho,  // real grid, size nx*ny*nz
    int n_atoms,
    int nx,
    int ny,
    int nz,
    float lx,
    float ly,
    float lz
) {
    size_t i0 = blockIdx.x * blockDim.x + threadIdx.x;
    size_t stride = blockDim.x * gridDim.x;

    int nxny = nx * ny;

    for (size_t i = i0; i < (size_t)n_atoms; i += stride) {
        float3 r = pos[i];

        float sx = r.x / lx * nx;
        float sy = r.y / ly * ny;
        float sz = r.z / lz * nz;

        int ix0, iy0, iz0;
        float wx[4], wy[4], wz[4];

        bspline4_weights(sx, &ix0, wx);
        bspline4_weights(sy, &iy0, wy);
        bspline4_weights(sz, &iz0, wz);

        float qi = q[i];

        for (int a=0; a<4; a++) {
            int ix = wrap(ix0 + a, nx);
            float wxa = wx[a];

            for (int b=0; b<4; b++) {
                int iy = wrap(iy0 + b, ny);
                float wxy = wxa * wy[b];

                int base = iy * nx + ix;

                for (int c=0; c<4; c++) {
                    int iz = wrap(iz0 + c, nz);
                    size_t idx = size_t(iz) * nxny + base;
                    atomicAdd(&rho[idx], qi * wxy * wz[c]);
                }
            }
        }
    }
}


// A kernel. Apply G(k) and gradient to get Exk/Eyk/Ezk
extern "C" __global__
void apply_ghat_and_grad(
    const float2* rho,
    float2* exk,
    float2* eyk,
    float2* ezk,
    //
    const float* kx,
    const float* ky,
    const float* kz,
    //
    const float* bx,
    const float* by,
    const float* bz,
    int nx, 
    int ny,
    int nz,
    float vol,
    float alpha,
    int n_real
 ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_cmplx = nx * ny * (nz/2 + 1);
    if (idx >= n_cmplx) return;

    int nxny = nx*ny;
    int iz = idx / nxny;          // 0 .. nz/2
    int rem = idx - iz * nxny;
    int iy = rem / nx;            // 0 .. ny-1
    int ix = rem - iy * nx;         // 0 .. nx-1

    float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];

    float k2  = fmaf(kxv, kxv, fmaf(kyv, kyv, kzv*kzv));
    if (k2 == 0.f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    float bmod2 = bx[ix] * by[iy] * bz[iz];
    if (bmod2 <= 1e-10f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    const float TWO_TAU = 12.56637061435917295385f; // 4π

    float ghat = (TWO_TAU / vol) * __expf(-k2 / (4.0f * alpha * alpha)) / (k2 * bmod2);
//     ghat *= float(n_real);   // compensate the 1/N you apply after the inverse

    float phi_k_real = rho[idx].x * ghat;
    float phi_k_im = rho[idx].y * ghat;

    // todo: Refactored; problem in a broken way.
    exk[idx].x = kxv * phi_k_real;
    exk[idx].y = kxv * phi_k_im;

    eyk[idx].x = kyv * phi_k_real;
    eyk[idx].y = kyv * phi_k_im;

    ezk[idx].x = kzv * phi_k_real;
    ezk[idx].y = kzv * phi_k_im;

    // after computing a,b and setting exk/eyk/ezk
    const bool rim_x = (ix==0) || ((nx%2)==0 && ix==(nx/2));
    const bool rim_y = (iy==0) || ((ny%2)==0 && iy==(ny/2));
    const bool rim_z = (iz==0) || ((nz%2)==0 && iz==(nz/2));

    // Imag parts must be zero on self-conjugate rims
    if (rim_x) exk[idx].y = 0.0f;
    if (rim_y) eyk[idx].y = 0.0f;
    if (rim_z) ezk[idx].y = 0.0f;
}


// todo: should these have the thread/stride splitting your main short-range kernes have??
extern "C" __global__
void gather_forces_to_atoms(
    const float3* pos,
    const float*  ex,
    const float*  ey,
    const float*  ez,
    const float*  q,
    float3*       out_f,
    int n_atoms,
    int nx,
    int ny,
    int nz,
    float lx,
    float ly,
    float lz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_atoms) return;

    float3 r = pos[i];
    float sx = r.x / lx * nx;
    float sy = r.y / ly * ny;
    float sz = r.z / lz * nz;

    int ix0 = __float2int_rd(sx) - 1;
    int iy0 = __float2int_rd(sy) - 1;
    int iz0 = __float2int_rd(sz) - 1;

    float u = sx - floorf(sx);
    float v = sy - floorf(sy);
    float w = sz - floorf(sz);

    float u2=u*u, u3=u2*u, um=1.f-u; float wx[4] = {(um*um*um)/6.f,(3.f*u3-6.f*u2+4.f)/6.f,(-3.f*u3+3.f*u2+3.f*u+1.f)/6.f,u3/6.f};
    float v2=v*v, v3=v2*v, vm=1.f-v; float wy[4] = {(vm*vm*vm)/6.f,(3.f*v3-6.f*v2+4.f)/6.f,(-3.f*v3+3.f*v2+3.f*v+1.f)/6.f,v3/6.f};
    float w2=w*w, w3=w2*w, wm=1.f-w; float wz[4] = {(wm*wm*wm)/6.f,(3.f*w3-6.f*w2+4.f)/6.f,(-3.f*w3+3.f*w2+3.f*w+1.f)/6.f,w3/6.f};

    float Exi=0.f, Eyi=0.f, Ezi=0.f;
    for (int a=0; a<4; a++){
        int ix = wrap(ix0 + a, nx);
        float wxa = wx[a];

        for (int b=0; b<4; b++){
            int iy = wrap(iy0 + b, ny);
            float wxy = wxa * wy[b];
            size_t base = size_t(iy)*nx + ix;

            for (int c=0; c<4; c++){
                int iz = wrap(iz0 + c, nz);
                float wfac = wxy * wz[c];
                size_t idx = size_t(iz)*nx*ny + base;

                Exi += wfac * ex[idx];
                Eyi += wfac * ey[idx];
                Ezi += wfac * ez[idx];
            }
        }
    }

    float s = q[i];
    out_f[i] = make_float3(Exi*s, Eyi*s, Ezi*s);
}


// A kernel
extern "C" __global__
void energy_half_spectrum(
    const float2* rho_k,
    const float* kx,
    const float* ky,
    const float* kz,
    const float* bx,
    const float* by,
    const float* bz,
    int nx,
    int ny,
    int nz,
    float vol,
    float alpha,
    double* out_partial
) {
    __shared__ double ssum[256];
    int tid = threadIdx.x;
    double acc = 0.0;

    int nxy = nx*ny;
    int n_cmplx = nxy*(nz/2 + 1);

    // CUFFT normalization for |rho_k|^2
    int N = nx*ny*nz;
    double invN2 = 1.0 / (double(N) * double(N));

    for (int idx = blockIdx.x*blockDim.x + tid; idx < n_cmplx; idx += gridDim.x*blockDim.x) {
        int iz = idx / nxy;
        int rem = idx - iz*nxy;
        int iy = rem / nx;
        int ix = rem - iy*nx;

        float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];
        float k2  = fmaf(kxv,kxv, fmaf(kyv,kyv, kzv*kzv));
        if (k2 == 0.f) continue;

        float bmod2 = bx[ix]*by[iy]*bz[iz];
        if (bmod2 <= 1e-10f) continue;

        float ghat = (2.0f*3.14159265358979323846f*2.0f / vol) * __expf(-k2/(4.0f*alpha*alpha)) / (k2*bmod2);

        float a = rho_k[idx].x, b = rho_k[idx].y;
        double mag2 = double(a)*a + double(b)*b;

        int twice = (iz==0 || ((nz%2)==0 && iz==(nz/2))) ? 1 : 2;
        acc += 0.5 * double(twice) * double(ghat) * (mag2 * invN2);
    }
    ssum[tid] = acc;
    __syncthreads();
    for (int s = blockDim.x/2; s>0; s>>=1) {
        if (tid < s) ssum[tid] += ssum[tid+s];
        __syncthreads();
    }
    if (tid==0) out_partial[blockIdx.x] = ssum[0];
}


// A utility kernel.
extern "C" __global__
void scale_vec(float* x, int n, float s) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) x[i] *= s;
}