// Contains code used by both GPU pipelines. This does not include a cuFFT dependency.

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

// Note: We use float2 instead of cufftComplex, as it's compatible with vkFFT.

__device__ __forceinline__ int wrap_i(int a, int n) { a %= n; return (a < 0) ? a + n : a; }

__global__ void scale3(float* ex, float* ey, float* ez, size_t n, float s) {
    size_t i = blockIdx.x * size_t(blockDim.x) + threadIdx.x;
    if (i < n) {
        ex[i] *= s; ey[i] *= s; ez[i] *= s;
    }
}

// Kernel for charge spreading
extern "C" __global__
void spread_charges(
    const float3* pos,
    const float*  q,
    float* rho,  // real grid, size nx*ny*nz
    int n_atoms,
    int nx,
    int ny,
    int nz,
    float lx,
    float ly,
    float lz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_atoms) return;

    float3 r = pos[i];
    float sx = r.x / lx * nx;
    float sy = r.y / ly * ny;
    float sz = r.z / lz * nz;

    int ix0 = __float2int_rd(sx) - 1;
    int iy0 = __float2int_rd(sy) - 1;
    int iz0 = __float2int_rd(sz) - 1;

    float u = sx - floorf(sx);
    float v = sy - floorf(sy);
    float w = sz - floorf(sz);

    float u2=u*u, u3=u2*u, um=1.f-u; float wx[4] = {(um*um*um)/6.f,(3.f*u3-6.f*u2+4.f)/6.f,(-3.f*u3+3.f*u2+3.f*u+1.f)/6.f,u3/6.f};
    float v2=v*v, v3=v2*v, vm=1.f-v; float wy[4] = {(vm*vm*vm)/6.f,(3.f*v3-6.f*v2+4.f)/6.f,(-3.f*v3+3.f*v2+3.f*v+1.f)/6.f,v3/6.f};
    float w2=w*w, w3=w2*w, wm=1.f-w; float wz[4] = {(wm*wm*wm)/6.f,(3.f*w3-6.f*w2+4.f)/6.f,(-3.f*w3+3.f*w2+3.f*w+1.f)/6.f,w3/6.f};

    float qi = q[i];

    for (int a=0; a<4; ++a) {
        int ix = wrap_i(ix0 + a, nx);
        float wxa = wx[a];
        for (int b=0; b<4; ++b) {
            int iy = wrap_i(iy0 + b, ny);
            float wxy = wxa * wy[b];
            size_t base = size_t(iy)*nx + ix;
            for (int c=0; c<4; ++c) {
                int iz = wrap_i(iz0 + c, nz);
                size_t idx = size_t(iz)*nx*ny + base;
                atomicAdd(&rho[idx], qi * wxy * wz[c]);
            }
        }
    }
}


// A kernel. Apply G(k) and gradient to get Exk/Eyk/Ezk
extern "C" __global__
void apply_ghat_and_grad(
    const float2* rho,
    float2* exk,
    float2* eyk,
    float2* ezk,
    const float* kx,
    const float* ky,
    const float* kz,
    const float* bx,
    const float* by,
    const float* bz,
    int nx, 
    int ny,
    int nz,
    float vol,
    float alpha
 ) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_cmplx = nx*ny*(nz/2 + 1);
    if (idx >= n_cmplx) return;

    int nxny = nx*ny;
    int iz = idx / nxny;          // 0 .. nz/2
    int rem = idx - iz*nxny;
    int iy = rem / nx;            // 0 .. ny-1
    int ix = rem - iy*nx;         // 0 .. nx-1

    float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];
    float k2  = fmaf(kxv, kxv, fmaf(kyv, kyv, kzv*kzv));
    if (k2 == 0.f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    float bmod2 = bx[ix] * by[iy] * bz[iz];
    if (bmod2 <= 1e-10f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    float ghat = (2.0f*3.14159265358979323846f*2.0f / vol) * __expf(-k2/(4.0f*alpha*alpha)) / (k2*bmod2);

    float a = rho[idx].x * ghat;
    float b = rho[idx].y * ghat;

    exk[idx].x =  kxv * b; exk[idx].y = -kxv * a;
    eyk[idx].x =  kyv * b; eyk[idx].y = -kyv * a;
    ezk[idx].x =  kzv * b; ezk[idx].y = -kzv * a;

    // after computing a,b and setting exk/eyk/ezk
    const bool rim_x = (ix==0) || ((nx%2)==0 && ix==(nx/2));
    const bool rim_y = (iy==0) || ((ny%2)==0 && iy==(ny/2));
    const bool rim_z = (iz==0) || ((nz%2)==0 && iz==(nz/2));
    // Imag parts must be zero on self-conjugate rims
    if (rim_x) exk[idx].y = 0.0f;
    if (rim_y) eyk[idx].y = 0.0f;
    if (rim_z) ezk[idx].y = 0.0f;
    // Optional (more conservative): zero the entire component on its Nyquist plane
    // if ((nx%2)==0 && ix==(nx/2)) exk[idx].x = exk[idx].y = 0.0f;
    // if ((ny%2)==0 && iy==(ny/2)) eyk[idx].x = eyk[idx].y = 0.0f;
    // if ((nz%2)==0 && iz==(nz/2)) ezk[idx].x = ezk[idx].y = 0.0f;
}


// todo: should these have the thread/stride splitting your main short-range kernes have??
extern "C" __global__
void gather_forces_to_atoms(
    const float3* pos,
    const float*  ex,
    const float*  ey,
    const float*  ez,
    const float*  q,
    float3*       out_f,
    int n_atoms,
    int nx,
    int ny,
    int nz,
    float lx,
    float ly,
    float lz
) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_atoms) return;

    float3 r = pos[i];
    float sx = r.x / lx * nx;
    float sy = r.y / ly * ny;
    float sz = r.z / lz * nz;

    int ix0 = __float2int_rd(sx) - 1;
    int iy0 = __float2int_rd(sy) - 1;
    int iz0 = __float2int_rd(sz) - 1;

    float u = sx - floorf(sx);
    float v = sy - floorf(sy);
    float w = sz - floorf(sz);

    float u2=u*u, u3=u2*u, um=1.f-u; float wx[4] = {(um*um*um)/6.f,(3.f*u3-6.f*u2+4.f)/6.f,(-3.f*u3+3.f*u2+3.f*u+1.f)/6.f,u3/6.f};
    float v2=v*v, v3=v2*v, vm=1.f-v; float wy[4] = {(vm*vm*vm)/6.f,(3.f*v3-6.f*v2+4.f)/6.f,(-3.f*v3+3.f*v2+3.f*v+1.f)/6.f,v3/6.f};
    float w2=w*w, w3=w2*w, wm=1.f-w; float wz[4] = {(wm*wm*wm)/6.f,(3.f*w3-6.f*w2+4.f)/6.f,(-3.f*w3+3.f*w2+3.f*w+1.f)/6.f,w3/6.f};

    float Exi=0.f, Eyi=0.f, Ezi=0.f;
    for (int a=0;a<4;++a){
        int ix = wrap_i(ix0 + a, nx);
        float wxa = wx[a];
        for (int b=0;b<4;++b){
            int iy = wrap_i(iy0 + b, ny);
            float wxy = wxa * wy[b];
            size_t base = size_t(iy)*nx + ix;
            for (int c=0;c<4;++c){
                int iz = wrap_i(iz0 + c, nz);
                float wfac = wxy * wz[c];
                size_t idx = size_t(iz)*nx*ny + base;
                Exi += wfac * ex[idx];
                Eyi += wfac * ey[idx];
                Ezi += wfac * ez[idx];
            }
        }
    }

    float s = q[i];
    out_f[i] = make_float3(Exi*s, Eyi*s, Ezi*s);
}



// A kernel
__global__ void energy_half_spectrum(
    const float2* rho_k,
    const float* kx,
    const float* ky,
    const float* kz,
    const float* bx,
    const float* by,
    const float* bz,
    int nx,
    int ny,
    int nz,
    float vol,
    float alpha,
    double* out_partial
) {
    extern __shared__ double ssum[];
    int tid = threadIdx.x;
    double acc = 0.0;

    int nxy = nx*ny;
    int n_cmplx = nxy*(nz/2 + 1);

    // CUFFT normalization for |rho_k|^2
    int N = nx*ny*nz;
    double invN2 = 1.0 / (double(N) * double(N));

    for (int idx = blockIdx.x*blockDim.x + tid; idx < n_cmplx; idx += gridDim.x*blockDim.x) {
        int iz = idx / nxy;
        int rem = idx - iz*nxy;
        int iy = rem / nx;
        int ix = rem - iy*nx;

        float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];
        float k2  = fmaf(kxv,kxv, fmaf(kyv,kyv, kzv*kzv));
        if (k2 == 0.f) continue;

        float bmod2 = bx[ix]*by[iy]*bz[iz];
        if (bmod2 <= 1e-10f) continue;

        float ghat = (2.0f*3.14159265358979323846f*2.0f / vol) * __expf(-k2/(4.0f*alpha*alpha)) / (k2*bmod2);

        float a = rho_k[idx].x, b = rho_k[idx].y;
        double mag2 = double(a)*a + double(b)*b;

        int twice = (iz==0 || ((nz%2)==0 && iz==(nz/2))) ? 1 : 2;
        acc += 0.5 * double(twice) * double(ghat) * (mag2 * invN2);
    }
    ssum[tid] = acc;
    __syncthreads();
    for (int s = blockDim.x/2; s>0; s>>=1) {
        if (tid < s) ssum[tid] += ssum[tid+s];
        __syncthreads();
    }
    if (tid==0) out_partial[blockIdx.x] = ssum[0];
}
