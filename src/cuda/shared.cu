// Contains code used by both GPU pipelines. This does not include a cuFFT dependency.

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cstdint>

// Note: We use float2 instead of cufftComplex, as it's compatible with vkFFT.

__device__ __forceinline__ int wrap_i(int a, int n) { a %= n; return (a < 0) ? a + n : a; }

__global__ void scale3(float* ex, float* ey, float* ez, size_t n, float s) {
    size_t i = blockIdx.x * size_t(blockDim.x) + threadIdx.x;
    if (i < n) {
        ex[i] *= s; ey[i] *= s; ez[i] *= s;
    }
}

// todo: Should this be marked Dvice?
extern "C" __global__
void scatter_rho_4x4x4(
    const float3* __restrict__ pos,
    const float*  __restrict__ q,
    float*        __restrict__ rho,   // real grid, size nx*ny*nz
    int n_atoms, int nx, int ny, int nz,
    float lx, float ly, float lz)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_atoms) return;

    float3 r = pos[i];
    float sx = r.x / lx * nx;
    float sy = r.y / ly * ny;
    float sz = r.z / lz * nz;

    int ix0 = __float2int_rd(sx) - 1;
    int iy0 = __float2int_rd(sy) - 1;
    int iz0 = __float2int_rd(sz) - 1;

    float u = sx - floorf(sx);
    float v = sy - floorf(sy);
    float w = sz - floorf(sz);

    float u2=u*u, u3=u2*u, um=1.f-u; float wx[4] = {(um*um*um)/6.f,(3.f*u3-6.f*u2+4.f)/6.f,(-3.f*u3+3.f*u2+3.f*u+1.f)/6.f,u3/6.f};
    float v2=v*v, v3=v2*v, vm=1.f-v; float wy[4] = {(vm*vm*vm)/6.f,(3.f*v3-6.f*v2+4.f)/6.f,(-3.f*v3+3.f*v2+3.f*v+1.f)/6.f,v3/6.f};
    float w2=w*w, w3=w2*w, wm=1.f-w; float wz[4] = {(wm*wm*wm)/6.f,(3.f*w3-6.f*w2+4.f)/6.f,(-3.f*w3+3.f*w2+3.f*w+1.f)/6.f,w3/6.f};

    float qi = q[i];

    for (int a=0; a<4; ++a) {
        int ix = wrap_i(ix0 + a, nx);
        float wxa = wx[a];
        for (int b=0; b<4; ++b) {
            int iy = wrap_i(iy0 + b, ny);
            float wxy = wxa * wy[b];
            size_t base = size_t(iy)*nx + ix;
            for (int c=0; c<4; ++c) {
                int iz = wrap_i(iz0 + c, nz);
                size_t idx = size_t(iz)*nx*ny + base;
                atomicAdd(&rho[idx], qi * wxy * wz[c]);
            }
        }
    }
}

extern "C"
void scatter_rho_4x4x4_launch(
    const void* pos, const void* q, void* rho,
    int n_atoms, int nx, int ny, int nz,
    float lx, float ly, float lz, void* cu_stream)
{
    auto s = reinterpret_cast<hipStream_t>(cu_stream);
    int threads = 256;
    int blocks  = (n_atoms + threads - 1) / threads;
    scatter_rho_4x4x4<<<blocks, threads, 0, s>>>(
        static_cast<const float3*>(pos),
        static_cast<const float*>(q),
        static_cast<float*>(rho),
        n_atoms, nx, ny, nz, lx, ly, lz);
}


// todo: Does this need a __device__ tag?
extern "C"
void scale_ExEyEz_after_c2r(float* ex, float* ey, float* ez,
                                 int nx, int ny, int nz, void* cu_stream) {
    auto s = reinterpret_cast<hipStream_t>(cu_stream);
    size_t n = size_t(nx)*ny*nz;
    int threads = 256;
    int blocks  = int((n + threads - 1) / threads);
    float invN  = 1.0f / float(n);
    scale3<<<blocks, threads, 0, s>>>(ex, ey, ez, n, invN);
}

// todo: Device tag required?
extern "C" __global__
void gather_forces_to_atoms(
    const float3* __restrict__ pos,
    const float*  __restrict__ ex,
    const float*  __restrict__ ey,
    const float*  __restrict__ ez,
    const float*  __restrict__ q,
    float3*       __restrict__ out_f,
    int n_atoms, int nx, int ny, int nz,
    float lx, float ly, float lz
    )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_atoms) return;

    float3 r = pos[i];
    float sx = r.x / lx * nx;
    float sy = r.y / ly * ny;
    float sz = r.z / lz * nz;

    int ix0 = __float2int_rd(sx) - 1;
    int iy0 = __float2int_rd(sy) - 1;
    int iz0 = __float2int_rd(sz) - 1;

    float u = sx - floorf(sx);
    float v = sy - floorf(sy);
    float w = sz - floorf(sz);

    float u2=u*u, u3=u2*u, um=1.f-u; float wx[4] = {(um*um*um)/6.f,(3.f*u3-6.f*u2+4.f)/6.f,(-3.f*u3+3.f*u2+3.f*u+1.f)/6.f,u3/6.f};
    float v2=v*v, v3=v2*v, vm=1.f-v; float wy[4] = {(vm*vm*vm)/6.f,(3.f*v3-6.f*v2+4.f)/6.f,(-3.f*v3+3.f*v2+3.f*v+1.f)/6.f,v3/6.f};
    float w2=w*w, w3=w2*w, wm=1.f-w; float wz[4] = {(wm*wm*wm)/6.f,(3.f*w3-6.f*w2+4.f)/6.f,(-3.f*w3+3.f*w2+3.f*w+1.f)/6.f,w3/6.f};

    float Exi=0.f, Eyi=0.f, Ezi=0.f;
    for (int a=0;a<4;++a){
        int ix = wrap_i(ix0 + a, nx);
        float wxa = wx[a];
        for (int b=0;b<4;++b){
            int iy = wrap_i(iy0 + b, ny);
            float wxy = wxa * wy[b];
            size_t base = size_t(iy)*nx + ix;
            for (int c=0;c<4;++c){
                int iz = wrap_i(iz0 + c, nz);
                float wfac = wxy * wz[c];
                size_t idx = size_t(iz)*nx*ny + base;
                Exi += wfac * ex[idx];
                Eyi += wfac * ey[idx];
                Ezi += wfac * ez[idx];
            }
        }
    }

    float s = q[i];
    out_f[i] = make_float3(Exi*s, Eyi*s, Ezi*s);
}

// todo: Does this need a __device__ tag?
extern "C"
void gather_forces_to_atoms_launch(
    const void* pos,
    const void* ex, const void* ey, const void* ez,
    const void* q,
    void* out_f,
    int n_atoms, int nx, int ny, int nz,
    float lx, float ly, float lz,
    float inv_n,
    void* cu_stream)
{
    auto s = reinterpret_cast<hipStream_t>(cu_stream);
    int threads = 256;
    int blocks  = (n_atoms + threads - 1) / threads;
    gather_forces_to_atoms<<<blocks, threads, 0, s>>>(
        static_cast<const float3*>(pos),
        static_cast<const float*>(ex),
        static_cast<const float*>(ey),
        static_cast<const float*>(ez),
        static_cast<const float*>(q),
        static_cast<float3*>(out_f),
        n_atoms, nx, ny, nz, lx, ly, lz
    );
}

extern "C" __global__
void apply_ghat_and_grad(
    const float2* __restrict__ rho,
    float2* __restrict__ exk,
    float2* __restrict__ eyk,
    float2* __restrict__ ezk,
    const float* __restrict__ kx,
    const float* __restrict__ ky,
    const float* __restrict__ kz,
    const float* __restrict__ bx,
    const float* __restrict__ by,
    const float* __restrict__ bz,
    int nx, int ny, int nz, float vol, float alpha)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int n_cmplx = nx*ny*(nz/2 + 1);
    if (idx >= n_cmplx) return;

    int nxny = nx*ny;
    int iz = idx / nxny;          // 0 .. nz/2
    int rem = idx - iz*nxny;
    int iy = rem / nx;            // 0 .. ny-1
    int ix = rem - iy*nx;         // 0 .. nx-1

    float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];
    float k2  = fmaf(kxv, kxv, fmaf(kyv, kyv, kzv*kzv));
    if (k2 == 0.f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    float bmod2 = bx[ix] * by[iy] * bz[iz];
    if (bmod2 <= 1e-10f) { exk[idx].x=exk[idx].y=0.f; eyk[idx]=exk[idx]; ezk[idx]=exk[idx]; return; }

    float ghat = (2.0f*3.14159265358979323846f*2.0f / vol) * __expf(-k2/(4.0f*alpha*alpha)) / (k2*bmod2);

    float a = rho[idx].x * ghat;
    float b = rho[idx].y * ghat;

    exk[idx].x =  kxv * b; exk[idx].y = -kxv * a;
    eyk[idx].x =  kyv * b; eyk[idx].y = -kyv * a;
    ezk[idx].x =  kzv * b; ezk[idx].y = -kzv * a;

    // after computing a,b and setting exk/eyk/ezk
    const bool rim_x = (ix==0) || ((nx%2)==0 && ix==(nx/2));
    const bool rim_y = (iy==0) || ((ny%2)==0 && iy==(ny/2));
    const bool rim_z = (iz==0) || ((nz%2)==0 && iz==(nz/2));
    // Imag parts must be zero on self-conjugate rims
    if (rim_x) exk[idx].y = 0.0f;
    if (rim_y) eyk[idx].y = 0.0f;
    if (rim_z) ezk[idx].y = 0.0f;
    // Optional (more conservative): zero the entire component on its Nyquist plane
    // if ((nx%2)==0 && ix==(nx/2)) exk[idx].x = exk[idx].y = 0.0f;
    // if ((ny%2)==0 && iy==(ny/2)) eyk[idx].x = eyk[idx].y = 0.0f;
    // if ((nz%2)==0 && iz==(nz/2)) ezk[idx].x = ezk[idx].y = 0.0f;
}

extern "C"
void apply_ghat_and_grad_launch(
    const void* rho,
    void* exk, void* eyk, void* ezk,
    const void* kx, const void* ky, const void* kz,
    const void* bx, const void* by, const void* bz,
    int nx, int ny, int nz, float vol, float alpha,
    void* cu_stream)
{
    auto s = reinterpret_cast<hipStream_t>(cu_stream);
    int n = nx*ny*(nz/2 + 1);          // <-- define n here

    int threads = 256;
    int blocks  = (n + threads - 1) / threads;

    apply_ghat_and_grad<<<blocks, threads, 0, s>>>(
        static_cast<const float2*>(rho),
        static_cast<float2*>(exk),
        static_cast<float2*>(eyk),
        static_cast<float2*>(ezk),

        static_cast<const float*>(kx),
        static_cast<const float*>(ky),
        static_cast<const float*>(kz),
        static_cast<const float*>(bx),
        static_cast<const float*>(by),
        static_cast<const float*>(bz),
        nx, ny, nz, vol, alpha
    );
}

__global__ void energy_half_spectrum(
    const float2* __restrict__ rho_k,
    const float* __restrict__ kx,
    const float* __restrict__ ky,
    const float* __restrict__ kz,
    const float* __restrict__ bx,
    const float* __restrict__ by,
    const float* __restrict__ bz,
    int nx, int ny, int nz, float vol, float alpha,
    double* __restrict__ out_partial)
{
    extern __shared__ double ssum[];
    int tid = threadIdx.x;
    double acc = 0.0;

    int nxy = nx*ny;
    int n_cmplx = nxy*(nz/2 + 1);

    // CUFFT normalization for |rho_k|^2
    int N = nx*ny*nz;
    double invN2 = 1.0 / (double(N) * double(N));

    for (int idx = blockIdx.x*blockDim.x + tid; idx < n_cmplx; idx += gridDim.x*blockDim.x) {
        int iz = idx / nxy;
        int rem = idx - iz*nxy;
        int iy = rem / nx;
        int ix = rem - iy*nx;

        float kxv = kx[ix], kyv = ky[iy], kzv = kz[iz];
        float k2  = fmaf(kxv,kxv, fmaf(kyv,kyv, kzv*kzv));
        if (k2 == 0.f) continue;

        float bmod2 = bx[ix]*by[iy]*bz[iz];
        if (bmod2 <= 1e-10f) continue;

        float ghat = (2.0f*3.14159265358979323846f*2.0f / vol) * __expf(-k2/(4.0f*alpha*alpha)) / (k2*bmod2);

        float a = rho_k[idx].x, b = rho_k[idx].y;
        double mag2 = double(a)*a + double(b)*b;

        int twice = (iz==0 || ((nz%2)==0 && iz==(nz/2))) ? 1 : 2;
        acc += 0.5 * double(twice) * double(ghat) * (mag2 * invN2);
    }
    ssum[tid] = acc;
    __syncthreads();
    for (int s = blockDim.x/2; s>0; s>>=1) {
        if (tid < s) ssum[tid] += ssum[tid+s];
        __syncthreads();
    }
    if (tid==0) out_partial[blockIdx.x] = ssum[0];
}

extern "C"
void energy_half_spectrum_launch(
    const void* rho_k,
    const void* kx, const void* ky, const void* kz,
    const void* bx, const void* by, const void* bz,
    int nx, int ny, int nz, float vol, float alpha,
    void* partial_sums,   // device ptr to double[blocks]
    int blocks, int threads, void* cu_stream)
{
    auto s = reinterpret_cast<hipStream_t>(cu_stream);
    size_t shmem = size_t(threads) * sizeof(double);

    energy_half_spectrum<<<blocks, threads, shmem, s>>>(
        static_cast<const float2*>(rho_k),
        static_cast<const float*>(kx), static_cast<const float*>(ky), static_cast<const float*>(kz),
        static_cast<const float*>(bx), static_cast<const float*>(by), static_cast<const float*>(bz),
        nx, ny, nz, vol, alpha,
        static_cast<double*>(partial_sums));
}